#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <getopt.h>
#include <assert.h>
#include <string.h>
/* OMP */
#include <omp.h>
/* CUDA */
#include <hip/hip_runtime.h>
/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
/* 3D vector manipulation abstraction */
#include <vec_gpu.h>

#ifndef MAX
#define MAX(x,y) ((x)>(y)?(x):(y))
#endif

#ifndef PI
#define PI 3.1415926535897
#endif

/** CUDA Error Handling Macros */
#define CUDA_ERROR_CHECK
#define CudaCheckError()	__cudaCheckError( __FILE__, __LINE__ )
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
	    printf("Error at %s:%d\n",__FILE__,__LINE__); \
	    return EXIT_FAILURE;}} while(0)





/** Window and view point */
typedef struct __Camera {
	Vec3 pos;
	double height, width;
} Camera;

/** CUDA Error Handling Util */
void __cudaCheckError( const char *file, const int line );

/** Util */
void save_grid(double * data, int px, int py, const char * fname);
void setArgs(int argc, char** argv);

/** Ray Tracer */
__global__ void rayTrace(int px, int py, int nrays, Camera camera, double * mat_grid);
__device__ void randomDirection(Vec3 * vptr);
__device__ void rand_double(double start, double end, double *result);

/** Global variables */
int num_pixel;
int num_ray;





/**************************************************
*
*	Driver
*
**************************************************/

int main(int argc, char ** argv) {

	num_pixel = 1000;
	num_ray = 1e7;
	Camera cam;

	cam.height = 20.0;
	cam.width = 20.0;

	setArgs(argc, argv);

	srand(time(NULL));

	/** Print GPU Device Name */
	hipDeviceProp_t prop;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);
	printf("%-25s%s\n", "CUDA Device: ", prop.name);

	/** Allocate serialized Grid Array on Host & Device */
	double * host_grid = (double*) malloc(num_pixel * num_pixel * sizeof(double));
	double * device_grid;
	CUDA_CALL(hipMalloc((void**) &device_grid, num_pixel*num_pixel*sizeof(double)));

	/** CUDA timer variables */
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float dt = 0;

	/** CUDA jobs */
	hipEventRecord(start, 0);
	/** Do something */
	rayTrace<<<1,1>>>(num_pixel, num_pixel, num_ray, cam, device_grid);
	/** Complete something */
	CudaCheckError();
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dt, start, stop);
	hipDeviceSynchronize();

	CUDA_CALL(hipMemcpy(host_grid, device_grid,
		num_pixel*num_pixel * sizeof(double), hipMemcpyDeviceToHost));

	save_grid(host_grid, num_pixel, num_pixel, "output.gpu.out");

	printf("%d\t%lf\n", num_pixel*num_pixel, dt/1000.0);

	free(host_grid);
	hipFree(device_grid);
}





/**************************************************
*
*	Ray tracer functions called on GPU by Host
*
**************************************************/

__global__ void rayTrace(int px, int py, int nrays, Camera camera, double * mat_grid) {

	/** Basic variables */
	int n;

	/** Model parameters */
	double radius = 6.0;
	Vec3 vec_c; // position of sphere centre
	Vec3 vec_l; // position of light source

	/** Ray tracer variables global */
	double dotp_cc;

	/** Window display */
	double w_max_x, w_min_x, w_max_z, w_min_z;

	/** Initialization */
	vec3(0.0, 12.0, 0.0, &vec_c);
	vec3(4.0, 4.0, -1.0, &vec_l);
	vec3(0.0, 10.0, 0.0, &camera.pos);
	w_max_x = camera.pos.x + camera.width * 0.5;
	w_min_x = camera.pos.x - camera.width * 0.5;
	w_max_z = camera.pos.z + camera.height * 0.5;
	w_min_z = camera.pos.z - camera.height * 0.5;
	vec3DotP(&vec_c, &vec_c, &dotp_cc);

	//omp_set_num_threads(16);
	//#pragma omp parallel for schedule(guided) shared(mat_grid) private(n)
	for (n=0; n<nrays; n++) {

		/** Ray tracer variables local */
		int i, j;
		double delta, solution, brightness;
		double dotp_vc;
		Vec3 vec_v; // view ray vector
		Vec3 vec_i; // position of intersection
		Vec3 vec_s; // direction of light source at I
		Vec3 vec_n; // unit normal vector at I
		Vec3 vec_w; // camera vector

		do { // sample random V from unit sphere
			do {
				randomDirection(&vec_v);
				vec3Scale(&vec_v, camera.pos.y / vec_v.y, &vec_w);
			} while (vec_w.x < w_min_x || vec_w.x > w_max_x || vec_w.z < w_min_z || vec_w.z > w_max_z);
			vec3DotP(&vec_v, &vec_c, &dotp_vc);
			delta = dotp_vc*dotp_vc + radius*radius - dotp_cc;
		} while (delta < 0); // delta > 0, enable to find an intersection

		solution = dotp_vc - sqrtf(delta);

		vec3Scale(&vec_v, solution, &vec_i);
		vec3Combine(&vec_i, &vec_c, 1.0, -1.0, &vec_n);
		vec3Normalize(&vec_n);
		vec3Combine(&vec_l, &vec_i, 1.0, -1.0, &vec_s);
		vec3Normalize(&vec_s);

		vec3DotP(&vec_s, &vec_n, &brightness);
		brightness = MAX(brightness,0);

		j = px - 1 - (int) ((double) px * (vec_w.x - w_min_x) / (camera.width));
		i = (int) ((double) py * (vec_w.z - w_min_z) / (camera.height));

		//#pragma omp atomic update
		mat_grid[i * px + j] += brightness;
		//atomicAdd(mat_grid[i * px + j], brightness);
	}
}

__device__ void randomDirection(Vec3 * vptr) {

	/*
	* return a unit vector of random direction.
	*/

	double angle_psi;
	double angle_theta;

	rand_double(0, 2 * PI, &angle_psi);
	rand_double(0, PI, &angle_theta);

	vptr->x = sin(angle_theta) * cos(angle_psi);
	vptr->y = sin(angle_theta) * sin(angle_psi);
	vptr->z = cos(angle_theta);
}

__device__ void rand_double(double start, double end, double *result) {

	/*
	* return a random double between start and end.
	*/

	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(0, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	/* hiprand works like rand - except that it takes a state as a parameter */
	double r = hiprand(&state) / (double) RAND_MAX;
	double x = (start < end) ? (start) : (end);
	double y = (start < end) ? (end) : (start);
	*result = (y - x) * r + x;
}

void __cudaCheckError(const char *file, const int line) {

	#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
	#endif
}





/**************************************************
*
*	Util functions called on Host
*
**************************************************/

void save_grid(double * data, int px, int py, const char * fname) {

	FILE * fp = fopen(fname, "w");
	int i, j;

	for (i=0; i<py; i++) {
		for (j=0; j<px; j++) {
			fprintf(fp, "%lf ", data[i * px + j]);
		} fprintf(fp, "\n");
	}

	fclose(fp);
}

void setArgs(int argc, char** argv) {

	/*
	* getopt_long stores the option index here.
	*/

	int option_index = 0;
	int ch;

	static struct option long_options[] = {
		//{"abc", 0|no_argument|required_argument|optional_argument, flag, 'a'},
		{"pixel", required_argument, 0, 'n'},
		{"nray", required_argument, 0, 'r'},
		{0, 0, 0, 0}
	};

	/* Detect the end of the options. */
	while ( (ch = getopt_long(argc, argv, "n:r:", long_options, &option_index)) != -1 ) {
		switch (ch) {
			case 'n':
				num_pixel = atoi(optarg);
				break;
			case 'r':
				num_ray = atoi(optarg);
				break;
			case '?':
				printf("Unknown option\n");
				break;
			case 0:
				break;
		}
	}
}
